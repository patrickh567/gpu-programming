#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_error.h"
#include "matAddKernel.h"

#include <stdio.h>

__global__ void matAddKernel(float* c, const float* y, const float* x, 
        const int N, const int M) {

    unsigned int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    if((tid_y < N) && (tid_x < M)) {
        c[tid_x + tid_y * M] = y[tid_x + tid_y * M] + x[tid_x + tid_y * M];
    }

}

int main(int argc, char* argv[]) {

    args_t args;
    parsArgs(&args, argc, argv);
    
    bool verify = args.verify;
    bool print = args.print;
    unsigned int numThreads_x = args.numThreads_x;
    unsigned int numThreads_y = args.numThreads_y;
    unsigned int M = args.M;
    unsigned int N = args.N;
    const unsigned maxElementSize = 10.0;
    
    // Cant assign more threads than there are elements in the matrix
    assert(numThreads_x <= M);
    assert(numThreads_y <= N);

    float* x = allocateMatrix(M, N);
    float* y = allocateMatrix(M, N);
    float* c = allocateMatrix(M, N);

    initializeMatrix(x, N, M, maxElementSize);
    initializeMatrix(y, N, M, maxElementSize);

    matAddWithCuda(c, x, y, N, M, verify, numThreads_x, numThreads_y);
    
    if(print) {
        printResults(c, x, y, N, M);
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    checkCuda(hipDeviceReset());

    free(x);
    free(y);
    
    return 0;

}

// Helper function for using CUDA to add vectors in parallel.
void matAddWithCuda(float* c, const float*  x, const float*  y, 
        const unsigned int N, const unsigned int M, const bool verify, 
        const unsigned int numThreads_x, const unsigned int numThreads_y) {

    float* dev_y;
    float* dev_x;
    float* dev_c;
    //hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    checkCuda(hipSetDevice(0));

    // Allocate GPU buffers for three matrices (two input, one output)    .
    checkCuda(hipMalloc((void**)&dev_y, M * N * sizeof(float)));
    checkCuda(hipMalloc((void**)&dev_x, M * N * sizeof(float)));
    checkCuda(hipMalloc((void**)&dev_c, M * N * sizeof(float)));
    //size_t pitch;
    //checkCuda(hipMallocPitch(&dev_x, &pitch, N, M));
    //printf("%i\n", (int)pitch);
    //checkCuda(hipMemcpy2D(dev_x, pitch, x, N*sizeof(float), M*sizeof(float), N, hipMemcpyHostToDevice));
    //exit(0);


    // Copy input vectors from host memory to GPU buffers.
    checkCuda(hipMemcpy(dev_y, y, M * N * sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(dev_x, x, M * N * sizeof(float), hipMemcpyHostToDevice));


    // Launch a kernel on the GPU with one thread for each element.
    dim3 dimGrid(ceil(((float)M)/((float)numThreads_x)), 
            ceil(((float)N)/((float)numThreads_y)), 1);

    dim3 dimBlock(numThreads_x, numThreads_y, 1);

    matAddKernel<<<dimGrid, dimBlock>>>(dev_c, dev_y, dev_x, N, M);

    // Check for any errors launching the kernel
    checkCuda(hipGetLastError());
    
    // Run CPU matrix addition, overlapping latency with CUDA matAdd computation
    float* cpu_c;
    if(verify) {
        cpu_c = (float*)malloc(sizeof(float)*M*N); 
        matAddWithCpu(cpu_c, x, y, N, M); 
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    checkCuda(hipDeviceSynchronize());

    // Copy output vector from GPU buffer to host memory.
    float* lin_c = (float*)malloc(M * N * sizeof(float));
    checkCuda(hipMemcpy(c, dev_c, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Verify CUDA computation is correct
    if(verify) {
        verifyCuda(cpu_c, c, M, N);
        free(cpu_c);
    }

    checkCuda(hipFree(dev_y));
    checkCuda(hipFree(dev_x));
    checkCuda(hipFree(dev_c));

}

void printMatrix(const float* c, const unsigned int N, const unsigned int M) {

    printf("{"); 
    for(int i = 0; i < N-1; i++) {
        printf("{");
        for(int j = 0; j < M-1; j++) {
            printf("%.2f, ", c[i * M + j]);
        }
        printf("%.2f},\n", c[i * M + M-1]);
    }
    printf("{");
    for(int j = 0; j < M-1; j++) {
        printf("%.2f, ", c[N-1 + j]);
    }
    printf("%.2f}}\n", c[N-1 + M-1]);

}

void printResults(const float*  c, const float* x, const float* y, 
        const unsigned int M, const unsigned int N) {
    
    printMatrix(x, N, M);
    printf("+\n");
    printMatrix(y, N, M);
    printf("=\n");
    printMatrix(c, N, M);

}

void matAddWithCpu(float* c, const float* x, const float*  y, 
        const unsigned int N, const unsigned int M) {

    for(int i = 0; i < N; i++) {
        for(int j = 0; j < M; j++) {
            c[i * M + j] = x[i * M + j] + y[i * M + j];
        }
    }

}

void verifyCuda(const float*  matAddCpu, const float* matAddCuda, 
        const unsigned int N, const unsigned int M) {

    const float tol = 1e-7; 
    bool error = false;
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < M; j++) {
            if(abs(matAddCpu[i * M + j] - matAddCuda[i * M + j]) > tol) {
                error = true;
                printf("CUDA matrix addition computation failed at index (%i, %i)\n", i, j);
                printf("CPU: %f, GPU: %f\n", matAddCpu[i * M + j], matAddCuda[i * M + j]);
                break;
            }
        }
    }
    if(error == false) {
        printf("CUDA matrix addition verification check passed.\n");
    }

}

float* allocateMatrix(const unsigned int M, const unsigned int N) {

    float* x = (float*)malloc(sizeof(float*)*N*M); 
    if(x == NULL) {
        printf("Malloc failed for array x.\n");
        exit(-1);
    }

    //for(int i = 0; i < M; i++) {
    //    x[i] = (float*)malloc(sizeof(float)*M); 
    //    if(x[i] == NULL) {
    //        printf("Malloc failed for array x.\n");
    //        exit(-1);
    //    }
    //}
    return x;

}

void initializeMatrix(float* x, const unsigned int N, const unsigned int M, 
        const unsigned int maxElementSize) {

    for(int i = 0; i < N; i++) {
        for(int j = 0; j < M; j++) {
            x[i * M + j] = (float)rand()/((float)RAND_MAX/maxElementSize); 
        }
    }

}

void parsArgs(args_t* args, int argc, char* argv[]) {

    if(argc > 7) {
        printf("Wrong number of args.\n");
        exit(1);
    }

    if(argc > 1) {
        for(int i = 1; i < argc; i++) {
            if(strcmp(argv[i], "--verify") == 0) {
                args->verify = true;
            } else if(strcmp(argv[i], "--print_results") == 0) {
                args->print = true;
            } else if(strcmp(argv[i], "--num_threads") == 0) {
                args->numThreads_x = atoi(argv[i+1]);
                args->numThreads_y = atoi(argv[i+1]);
                i++;
            } else if(strcmp(argv[i], "--rows") == 0) {
                args->N = atoi(argv[i+1]);
                i++;
            } else if(strcmp(argv[i], "--cols") == 0) {
                args->M = atoi(argv[i+1]);
                i++;
            } else {
                printf("Unrecognized arg. Aborting\n"); 
                exit(1);
            }
        }
    }

}
